#include "CUDA_WRAPPER.h"

#include "hip/hip_runtime.h"
#include ""

bool allocateCUDAFloatBuffer(float* ptr, float** dev_ptr, size_t numFloats)
{
	hipError_t deviceStatus;

	deviceStatus = hipMalloc((void**)dev_ptr, numFloats * sizeof(float));

	if (!checkCudaErrorStatus(deviceStatus, "hipMalloc"))
	{
		return false;
	}
	else
	{
		return true;
	}
}

bool freeCUDAFloatBuffer(float* dev_ptr)
{
	hipError_t deviceStatus;

	deviceStatus = hipFree(dev_ptr);

	if (!checkCudaErrorStatus(deviceStatus, "hipMalloc"))
	{
		return false;
	}
	else
	{
		return true;
	}
}

bool copyMem2Device(float* dev_ptr, float* ptr, size_t numFloats)
{
	hipError_t deviceStatus;

	deviceStatus = hipMemcpy(dev_ptr, ptr, numFloats * sizeof(float), hipMemcpyHostToDevice);

	if (!checkCudaErrorStatus(deviceStatus, "hipMemcpy (hipMemcpyHostToDevice)"))
	{
		return false;
	}
	else
	{
		return true;
	}
}

bool copyMemFromDevice(float* ptr, float* dev_ptr, size_t numFloats)
{
	hipError_t deviceStatus;

	deviceStatus = hipMemcpy(ptr, dev_ptr, numFloats * sizeof(float), hipMemcpyDeviceToHost);

	if (!checkCudaErrorStatus(deviceStatus, "hipMemcpy (hipMemcpyDeviceToHost)"))
	{
		return false;
	}
	else
	{
		return true;
	}
}