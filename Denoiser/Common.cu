#include "CUDA_WRAPPER.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>

bool checkCudaErrorStatus(hipError_t status, const std::string& functionName)
{
	if (status != hipSuccess)
	{
		std::cerr << "ERROR [ " << functionName << " ]: " << hipGetErrorString(status) << std::endl;
		return false;
	}
	return true;
}

bool getCudaDeviceProperties(int device)
{
	hipError_t deviceStatus;
	hipDeviceProp_t properties;
	deviceStatus = hipGetDeviceProperties(&properties, device);

	if (!checkCudaErrorStatus(deviceStatus, "hipGetDeviceProperties"))
	{
		return false;
	}

	std::cout << "Compute Capabilities for " << properties.name << " : " << std::endl;
	std::cout << "Major: " << properties.major << ", Minor: " << properties.minor << std::endl;
	std::cout << "Details: " << std::endl;
	std::cout << "	Num of SM    : " << properties.multiProcessorCount << std::endl;
	std::cout << "	Mem per Block: " << properties.sharedMemPerBlock << std::endl;
	std::cout << "	Mem per SM   : " << properties.sharedMemPerMultiprocessor << std::endl;

	return true;
}

bool queryCUDADevices()
{
	hipError_t deviceStatus;

	int deviceCount = 0;
	deviceStatus = hipGetDeviceCount(&deviceCount);

	if (!checkCudaErrorStatus(deviceStatus, "hipGetDeviceCount"))
	{
		return false;
	}

	std::cout << "Num CUDA Devices Found: " << deviceCount << std::endl;

	return true;
}

bool startCUDAApplication(int device)
{
	hipError_t deviceStatus;

	deviceStatus = hipSetDevice(device);
	
	if (!checkCudaErrorStatus(deviceStatus, "hipSetDevice"))
	{
		return false;
	}
	else
	{
		return true;
	}
}

bool quiteCUDAApplication()
{
	hipError_t deviceStatus;

	deviceStatus = deviceStatus = hipDeviceReset();

	if (!checkCudaErrorStatus(deviceStatus, "hipDeviceReset"))
	{
		return false;
	}
	else
	{
		return true;
	}
}